#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 92 // blocks
#define K 32 // threads

// jsf32 random
typedef uint32_t u4;
typedef struct ranctx { u4 a; u4 b; u4 c; u4 d; } ranctx;

#define rot32(x,k) (((x)<<(k))|((x)>>(32-(k))))
__device__ u4 ranval( ranctx *x ) {
    u4 e = x->a - rot32(x->b, 27);
    x->a = x->b ^ rot32(x->c, 17);
    x->b = x->c + x->d;
    x->c = x->d + e;
    x->d = e + x->a;
    return x->d;
}

__device__ void raninit( ranctx *x, u4 seed ) {
    u4 i;
    x->a = 0xf1ea5eed, x->b = x->c = x->d = seed;
    for (i=0; i<20; ++i) {
        (void)ranval(x);
    }
}

u4 ranval_normal( ranctx *x ) {
    u4 e = x->a - rot32(x->b, 27);
    x->a = x->b ^ rot32(x->c, 17);
    x->b = x->c + x->d;
    x->c = x->d + e;
    x->d = e + x->a;
    return x->d;
}

void raninit_normal( ranctx *x, u4 seed ) {
    u4 i;
    x->a = 0xf1ea5eed, x->b = x->c = x->d = seed;
    for (i=0; i<20; ++i) {
        (void)ranval_normal(x);
    }
}

__device__ void md5_hash_m1(uint32_t* m, uint32_t output[4], uint32_t s[4]) {
    // init
    uint32_t state[4];
    for (int i = 0; i < 4; i++) {
        state[i] = s[i];
    }

    uint32_t A;
    uint32_t B;
    uint32_t C;
    uint32_t D;

    uint32_t AOld;
    uint32_t BOld;
    uint32_t COld;
    uint32_t DOld;

    uint32_t ANew;
    uint32_t BNew;
    uint32_t CNew;
    uint32_t DNew;


#define S(x,n)                                                          \
            ( ( (x) << (n) ) | ( ( (x) & 0xFFFFFFFF) >> ( 32 - (n) ) ) )

#define P(a,b,c,d,k,s,t)                                                \
            do                                                                  \
            {                                                                   \
                (a) += F((b),(c),(d)) + m[(k)] + (t);                     \
                (a) = S((a),(s)) + (b);                                         \
            } while( 0 )

    A = state[0];
    B = state[1];
    C = state[2];
    D = state[3];

    // 16 kroków, jedno F, jeden cykl
#define F(x,y,z) ((z) ^ ((x) & ((y) ^ (z))))
    AOld = A;
    P( A, B, C, D,  0,  7, 0xD76AA478 );
        ANew = (A & 0x71dff7df) | 0x84200000;
        // after applying changes
        if (A != ANew) { // need to compute m1'
            m[0] = ((((ANew - B)) >> (7)) | ((((ANew - B) ) & 0xFFFFFFFF) << (32 - (7))))
                   - AOld - F(B, C, D) - 0xD76AA478;
            A = ANew;
        }

    DOld = D;
    P( D, A, B, C,  1, 12, 0xE8C7B756 );

        DNew = (D & 0x1c06719) | 0x8c000800 | (A & 0x701f10c0);
        if (D != DNew) { // need to compute m1'
            m[1] = ((((DNew - A)) >> (12)) | ((((DNew - A) ) & 0xFFFFFFFF) << (32 - (12))))
                   - DOld - F(A, B, C) - 0xE8C7B756;
            D = DNew;
        }

    COld = C;
    P( C, D, A, B,  2, 17, 0x242070DB );

        CNew = (C & 0x1c0e601) | 0xbe1f0966 | (D & 0x00000018);
        if (C != CNew) { // need to compute m1'
            m[2] = ((((CNew - D)) >> (17)) | ((((CNew - D) ) & 0xFFFFFFFF) << (32 - (17))))
                   - COld - F(D, A, B) - 0x242070DB;
            C = CNew;
        }

    BOld = B;
    P( B, C, D, A,  3, 22, 0xC1BDCEEE );

        BNew = (B & 0x1c0e000) | 0xba040010 | (C & 0x00000601);
        if (B != BNew) { // need to compute m1'
            m[3] = ((((BNew - C)) >> (22)) | ((((BNew - C) ) & 0xFFFFFFFF) << (32 - (22))))
                   - BOld - F(C, D, A) - 0xC1BDCEEE;
            B = BNew;
        }

    AOld = A;
    P( A, B, C, D,  4,  7, 0xF57C0FAF );

        ANew = (A & 0x3c0e000) | 0x482f0e50;
        if (A != ANew) { // need to compute m1'
            m[4] = ((((ANew - B)) >> (7)) | ((((ANew - B) ) & 0xFFFFFFFF) << (32 - (7))))
                   - AOld - F(B, C, D) - 0xF57C0FAF;
            A = ANew;
        }

    DOld = D;
    P( D, A, B, C,  5, 12, 0x4787C62A );

        DNew = (D & 0x61cce000) | 0x04220c56;
        if (D != DNew) { // need to compute m1'
            m[5] = ((((DNew - A)) >> (12)) | ((((DNew - A) ) & 0xFFFFFFFF) << (32 - (12))))
                   - DOld - F(A, B, C) - 0x4787C62A;
            D = DNew;
        }

    COld = C;
    P( C, D, A, B,  6, 17, 0xA8304613 );

        CNew = (C & 0x604c603e) | 0x96011e01 | (D & 0x01808000);
        if (C != CNew) { // need to compute m1'
            m[6] = ((((CNew - D)) >> (17)) | ((((CNew - D) ) & 0xFFFFFFFF) << (32 - (17))))
                   - COld - F(D, A, B) - 0xA8304613;
            C = CNew;
        }

    BOld = B;
    P( B, C, D, A,  7, 22, 0xFD469501 );

        BNew = (B & 0x604c7c3c) | 0x843283c0 | (C & 0x00000002);
        if (B != BNew) { // need to compute m1'
            m[7] = ((((BNew - C)) >> (22)) | ((((BNew - C) ) & 0xFFFFFFFF) << (32 - (22))))
                   - BOld - F(C, D, A) - 0xFD469501;
            B = BNew;
        }

    AOld = A;
    P( A, B, C, D,  8,  7, 0x698098D8 );
        ANew = (A & 0x607c6c3c) | 0x9c0101c1 | (B & 0x00001000);
        if (A != ANew) { // need to compute m1'
            m[8] = ((((ANew - B)) >> (7)) | ((((ANew - B) ) & 0xFFFFFFFF) << (32 - (7))))
                   - AOld - F(B, C, D) - 0x698098D8;
            A = ANew;
        }

    DOld = D;
    P( D, A, B, C,  9, 12, 0x8B44F7AF );
        DNew = (D &  0x78786c3c) | 0x878383c0;
        if (D != DNew) { // need to compute m1'
            m[9] = ((((DNew - A)) >> (12)) | ((((DNew - A) ) & 0xFFFFFFFF) << (32 - (12))))
                   - DOld - F(A, B, C) - 0x8B44F7AF;
            D = DNew;
        }

    COld = C;
    P( C, D, A, B, 10, 17, 0xFFFF5BB1 );

        CNew = (C & 0x7ff00c3c) | 0x800583c3 | (D & 0x86000);
        if (C != CNew) { // need to compute m1'
            m[10] = ((((CNew - D)) >> (17)) | ((((CNew - D) ) & 0xFFFFFFFF) << (32 - (17))))
                    - COld - F(D, A, B) - 0xFFFF5BB1;
            C = CNew;
        }

    BOld = B;
    P( B, C, D, A, 11, 22, 0x895CD7BE );

        BNew = (B & 0xf00f7f) | 0x80081080 | (C & 0x7f000000);
        if (B != BNew) { // need to compute m1'
            m[11] = ((((BNew - C)) >> (22)) | ((((BNew - C) ) & 0xFFFFFFFF) << (32 - (22))))
                    - BOld - F(C, D, A) - 0x895CD7BE;
            B = BNew;
        }

    AOld = A;
    P( A, B, C, D, 12,  7, 0x6B901122 );

        ANew = (A & 0xf01f77) | 0x3f0fe008;
        if (A != ANew) { // need to compute m1'
            m[12] = ((((ANew - B)) >> (7)) | ((((ANew - B) ) & 0xFFFFFFFF) << (32 - (7))))
                    - AOld - F(B, C, D) - 0x6B901122;
            A = ANew;
        }

    DOld = D;
    P( D, A, B, C, 13, 12, 0xFD987193 );

        DNew = (D & 0xf01f77) | 0x400be088;
        if (D != DNew) { // need to compute m1'
            m[13] = ((((DNew - A)) >> (12)) | ((((DNew - A) ) & 0xFFFFFFFF) << (32 - (12))))
                    - DOld - F(A, B, C) - 0xFD987193;
            D = DNew;
        }

    COld = C;
    P( C, D, A, B, 14, 17, 0xA679438E );

        CNew = (C & 0xff7ff7) | 0x7d000000;
        if (C != CNew) { // need to compute m1'
            m[14] = ((((CNew - D)) >> (17)) | ((((CNew - D) ) & 0xFFFFFFFF) << (32 - (17))))
                    - COld - F(D, A, B) - 0xA679438E;
            C = CNew;
        }

    BOld = B;
    P( B, C, D, A, 15, 22, 0x49B40821 );

        BNew = (B & 0x5fffffff) | 0x20000000;
        if (B != BNew) { // need to compute m1'
            m[15] = ((((BNew - C)) >> (22)) | ((((BNew - C) ) & 0xFFFFFFFF) << (32 - (22))))
                    - BOld - F(C, D, A) - 0x49B40821;
            B = BNew;
        }
#undef F

#define F(x,y,z) ((y) ^ ((z) & ((x) ^ (y))))
    P( A, B, C, D,  1,  5, 0xF61E2562 );
    P( D, A, B, C,  6,  9, 0xC040B340 );
    P( C, D, A, B, 11, 14, 0x265E5A51 );
    P( B, C, D, A,  0, 20, 0xE9B6C7AA );
    P( A, B, C, D,  5,  5, 0xD62F105D );
    P( D, A, B, C, 10,  9, 0x02441453 );
    P( C, D, A, B, 15, 14, 0xD8A1E681 );
    P( B, C, D, A,  4, 20, 0xE7D3FBC8 );
    P( A, B, C, D,  9,  5, 0x21E1CDE6 );
    P( D, A, B, C, 14,  9, 0xC33707D6 );
    P( C, D, A, B,  3, 14, 0xF4D50D87 );
    P( B, C, D, A,  8, 20, 0x455A14ED );
    P( A, B, C, D, 13,  5, 0xA9E3E905 );
    P( D, A, B, C,  2,  9, 0xFCEFA3F8 );
    P( C, D, A, B,  7, 14, 0x676F02D9 );
    P( B, C, D, A, 12, 20, 0x8D2A4C8A );

#undef F

#define F(x,y,z) ((x) ^ (y) ^ (z))
    P( A, B, C, D,  5,  4, 0xFFFA3942 );
    P( D, A, B, C,  8, 11, 0x8771F681 );
    P( C, D, A, B, 11, 16, 0x6D9D6122 );
    P( B, C, D, A, 14, 23, 0xFDE5380C );
    P( A, B, C, D,  1,  4, 0xA4BEEA44 );
    P( D, A, B, C,  4, 11, 0x4BDECFA9 );
    P( C, D, A, B,  7, 16, 0xF6BB4B60 );
    P( B, C, D, A, 10, 23, 0xBEBFBC70 );
    P( A, B, C, D, 13,  4, 0x289B7EC6 );
    P( D, A, B, C,  0, 11, 0xEAA127FA );
    P( C, D, A, B,  3, 16, 0xD4EF3085 );
    P( B, C, D, A,  6, 23, 0x04881D05 );
    P( A, B, C, D,  9,  4, 0xD9D4D039 );
    P( D, A, B, C, 12, 11, 0xE6DB99E5 );
    P( C, D, A, B, 15, 16, 0x1FA27CF8 );
    P( B, C, D, A,  2, 23, 0xC4AC5665 );
#undef F

#define F(x,y,z) ((y) ^ ((x) | ~(z)))
    P( A, B, C, D,  0,  6, 0xF4292244 );
    P( D, A, B, C,  7, 10, 0x432AFF97 );
    P( C, D, A, B, 14, 15, 0xAB9423A7 );
    P( B, C, D, A,  5, 21, 0xFC93A039 );
    P( A, B, C, D, 12,  6, 0x655B59C3 );
    P( D, A, B, C,  3, 10, 0x8F0CCC92 );
    P( C, D, A, B, 10, 15, 0xFFEFF47D );
    P( B, C, D, A,  1, 21, 0x85845DD1 );
    P( A, B, C, D,  8,  6, 0x6FA87E4F );
    P( D, A, B, C, 15, 10, 0xFE2CE6E0 );
    P( C, D, A, B,  6, 15, 0xA3014314 );
    P( B, C, D, A, 13, 21, 0x4E0811A1 );
    P( A, B, C, D,  4,  6, 0xF7537E82 );
    P( D, A, B, C, 11, 10, 0xBD3AF235 );
    P( C, D, A, B,  2, 15, 0x2AD7D2BB );
    P( B, C, D, A,  9, 21, 0xEB86D391 );
#undef F

    state[0] += A;
    state[1] += B;
    state[2] += C;
    state[3] += D;

    output[0] = state[0];
    output[1] = state[1];
    output[2] = state[2];
    output[3] = state[3];
}

__device__ void md5_hash(uint32_t* m, uint32_t output[4], uint32_t s[4]) {
    // init
    uint32_t state[4];
    for (int i = 0; i < 4; i++) {
        state[i] = s[i];
    }

    uint32_t A;
    uint32_t B;
    uint32_t C;
    uint32_t D;


#define S(x,n)                                                          \
            ( ( (x) << (n) ) | ( ( (x) & 0xFFFFFFFF) >> ( 32 - (n) ) ) )

#define P(a,b,c,d,k,s,t)                                                \
            do                                                                  \
            {                                                                   \
                (a) += F((b),(c),(d)) + m[(k)] + (t);                     \
                (a) = S((a),(s)) + (b);                                         \
            } while( 0 )

    A = state[0];
    B = state[1];
    C = state[2];
    D = state[3];

    // 16 kroków, jedno F, jeden cykl
#define F(x,y,z) ((z) ^ ((x) & ((y) ^ (z))))
    P( A, B, C, D,  0,  7, 0xD76AA478 );
    P( D, A, B, C,  1, 12, 0xE8C7B756 );
    P( C, D, A, B,  2, 17, 0x242070DB );
    P( B, C, D, A,  3, 22, 0xC1BDCEEE );
    P( A, B, C, D,  4,  7, 0xF57C0FAF );
    P( D, A, B, C,  5, 12, 0x4787C62A );
    P( C, D, A, B,  6, 17, 0xA8304613 );
    P( B, C, D, A,  7, 22, 0xFD469501 );
    P( A, B, C, D,  8,  7, 0x698098D8 );
    P( D, A, B, C,  9, 12, 0x8B44F7AF );
    P( C, D, A, B, 10, 17, 0xFFFF5BB1 );
    P( B, C, D, A, 11, 22, 0x895CD7BE );
    P( A, B, C, D, 12,  7, 0x6B901122 );
    P( D, A, B, C, 13, 12, 0xFD987193 );
    P( C, D, A, B, 14, 17, 0xA679438E );
    P( B, C, D, A, 15, 22, 0x49B40821 );
#undef F

#define F(x,y,z) ((y) ^ ((z) & ((x) ^ (y))))
    P( A, B, C, D,  1,  5, 0xF61E2562 );
    P( D, A, B, C,  6,  9, 0xC040B340 );
    P( C, D, A, B, 11, 14, 0x265E5A51 );
    P( B, C, D, A,  0, 20, 0xE9B6C7AA );
    P( A, B, C, D,  5,  5, 0xD62F105D );
    P( D, A, B, C, 10,  9, 0x02441453 );
    P( C, D, A, B, 15, 14, 0xD8A1E681 );
    P( B, C, D, A,  4, 20, 0xE7D3FBC8 );
    P( A, B, C, D,  9,  5, 0x21E1CDE6 );
    P( D, A, B, C, 14,  9, 0xC33707D6 );
    P( C, D, A, B,  3, 14, 0xF4D50D87 );
    P( B, C, D, A,  8, 20, 0x455A14ED );
    P( A, B, C, D, 13,  5, 0xA9E3E905 );
    P( D, A, B, C,  2,  9, 0xFCEFA3F8 );
    P( C, D, A, B,  7, 14, 0x676F02D9 );
    P( B, C, D, A, 12, 20, 0x8D2A4C8A );

#undef F

#define F(x,y,z) ((x) ^ (y) ^ (z))
    P( A, B, C, D,  5,  4, 0xFFFA3942 );
    P( D, A, B, C,  8, 11, 0x8771F681 );
    P( C, D, A, B, 11, 16, 0x6D9D6122 );
    P( B, C, D, A, 14, 23, 0xFDE5380C );
    P( A, B, C, D,  1,  4, 0xA4BEEA44 );
    P( D, A, B, C,  4, 11, 0x4BDECFA9 );
    P( C, D, A, B,  7, 16, 0xF6BB4B60 );
    P( B, C, D, A, 10, 23, 0xBEBFBC70 );
    P( A, B, C, D, 13,  4, 0x289B7EC6 );
    P( D, A, B, C,  0, 11, 0xEAA127FA );
    P( C, D, A, B,  3, 16, 0xD4EF3085 );
    P( B, C, D, A,  6, 23, 0x04881D05 );
    P( A, B, C, D,  9,  4, 0xD9D4D039 );
    P( D, A, B, C, 12, 11, 0xE6DB99E5 );
    P( C, D, A, B, 15, 16, 0x1FA27CF8 );
    P( B, C, D, A,  2, 23, 0xC4AC5665 );
#undef F

#define F(x,y,z) ((y) ^ ((x) | ~(z)))
    P( A, B, C, D,  0,  6, 0xF4292244 );
    P( D, A, B, C,  7, 10, 0x432AFF97 );
    P( C, D, A, B, 14, 15, 0xAB9423A7 );
    P( B, C, D, A,  5, 21, 0xFC93A039 );
    P( A, B, C, D, 12,  6, 0x655B59C3 );
    P( D, A, B, C,  3, 10, 0x8F0CCC92 );
    P( C, D, A, B, 10, 15, 0xFFEFF47D );
    P( B, C, D, A,  1, 21, 0x85845DD1 );
    P( A, B, C, D,  8,  6, 0x6FA87E4F );
    P( D, A, B, C, 15, 10, 0xFE2CE6E0 );
    P( C, D, A, B,  6, 15, 0xA3014314 );
    P( B, C, D, A, 13, 21, 0x4E0811A1 );
    P( A, B, C, D,  4,  6, 0xF7537E82 );
    P( D, A, B, C, 11, 10, 0xBD3AF235 );
    P( C, D, A, B,  2, 15, 0x2AD7D2BB );
    P( B, C, D, A,  9, 21, 0xEB86D391 );
#undef F

    state[0] += A;
    state[1] += B;
    state[2] += C;
    state[3] += D;

    output[0] = state[0];
    output[1] = state[1];
    output[2] = state[2];
    output[3] = state[3];
}

__global__ void attack(uint32_t seed, int* success) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
   // success[idx] = 0;
    // m0 hashed (initial state for m1)
//    uint32_t s1[] = {0x52589324, 0x3093d7ca, 0x2a06dc54, 0x20c5be06};
    // m0' hashed (initial state for m1')
//    uint32_t s2[] = {0xd2589324, 0xb293d7ca, 0xac06dc54, 0xa2c5be06};
	uint32_t s1[] = {0xfec19f38, 0xf430a8ea, 0x8ab480e4, 0x58892490};
    uint32_t s2[] = {0x7ec19f38, 0x7630a8ea, 0xcb480e4, 0xda892490};
    uint32_t m1[16];
    uint32_t m1_added[16];
    uint32_t digest1[4];
    uint32_t digest2[4];
    int equal = 0;
    // initialize random generator
    struct ranctx state;
    raninit(&state, seed + idx);
    // one thread performs 2^20 experiments
    for (int j = 0; j < 1048576 ; j++) {
        // get random message m1
        for (int i = 0; i < 16; i++) {
            m1[i] = ranval(&state);
       //     d_messages[idx + i] = m1[i];
        }

        // modify and check if this works
        md5_hash_m1(m1, digest1, s1);

        // construct m1'
        for (int i = 0; i < 16; i++) {
            m1_added[i] = m1[i];
        }
        m1_added[4] += 0x80000000;
        m1_added[11] -= 0x00008000;
        m1_added[14] += 0x80000000;

        md5_hash(m1_added, digest2, s2);
       // printf("\n%x -------------------------------\n", digest2[0]);

        // check if hashes equal
        equal = 1;
        for (int i = 0; i < 4; i++) {
            if (digest1[i] != digest2[i]) {
                equal = 0;
            }
        }
        if (equal == 1) { // collision found
            success[idx] = 1;
            break;
        }
    }
}

void md5_hash_normal(uint32_t* m, uint32_t output[4], uint32_t* s) {
    uint32_t state[4];
    for (int i = 0; i < 4; i++) {
        state[i] = s[i];
    }

    uint32_t A;
    uint32_t B;
    uint32_t C;
    uint32_t D;

#define S(x,n)                                                          \
            ( ( (x) << (n) ) | ( ( (x) & 0xFFFFFFFF) >> ( 32 - (n) ) ) )

#define P(a,b,c,d,k,s,t)                                                \
            do                                                                  \
            {                                                                   \
                (a) += F((b),(c),(d)) + m[(k)] + (t);                     \
                (a) = S((a),(s)) + (b);                                         \
            } while( 0 )

    A = state[0];
    B = state[1];
    C = state[2];
    D = state[3];

    // 16 kroków, jedno F, jeden cykl
#define F(x,y,z) ((z) ^ ((x) & ((y) ^ (z))))
    P( A, B, C, D,  0,  7, 0xD76AA478 );
    P( D, A, B, C,  1, 12, 0xE8C7B756 );
    P( C, D, A, B,  2, 17, 0x242070DB );
    P( B, C, D, A,  3, 22, 0xC1BDCEEE );
    P( A, B, C, D,  4,  7, 0xF57C0FAF );
    P( D, A, B, C,  5, 12, 0x4787C62A );
    P( C, D, A, B,  6, 17, 0xA8304613 );
    P( B, C, D, A,  7, 22, 0xFD469501 );
    P( A, B, C, D,  8,  7, 0x698098D8 );
    P( D, A, B, C,  9, 12, 0x8B44F7AF );
    P( C, D, A, B, 10, 17, 0xFFFF5BB1 );
    P( B, C, D, A, 11, 22, 0x895CD7BE );
    P( A, B, C, D, 12,  7, 0x6B901122 );
    P( D, A, B, C, 13, 12, 0xFD987193 );
    P( C, D, A, B, 14, 17, 0xA679438E );
    P( B, C, D, A, 15, 22, 0x49B40821 );
#undef F

#define F(x,y,z) ((y) ^ ((z) & ((x) ^ (y))))
    P( A, B, C, D,  1,  5, 0xF61E2562 );
    P( D, A, B, C,  6,  9, 0xC040B340 );
    P( C, D, A, B, 11, 14, 0x265E5A51 );
    P( B, C, D, A,  0, 20, 0xE9B6C7AA );
    P( A, B, C, D,  5,  5, 0xD62F105D );
    P( D, A, B, C, 10,  9, 0x02441453 );
    P( C, D, A, B, 15, 14, 0xD8A1E681 );
    P( B, C, D, A,  4, 20, 0xE7D3FBC8 );
    P( A, B, C, D,  9,  5, 0x21E1CDE6 );
    P( D, A, B, C, 14,  9, 0xC33707D6 );
    P( C, D, A, B,  3, 14, 0xF4D50D87 );
    P( B, C, D, A,  8, 20, 0x455A14ED );
    P( A, B, C, D, 13,  5, 0xA9E3E905 );
    P( D, A, B, C,  2,  9, 0xFCEFA3F8 );
    P( C, D, A, B,  7, 14, 0x676F02D9 );
    P( B, C, D, A, 12, 20, 0x8D2A4C8A );

#undef F

#define F(x,y,z) ((x) ^ (y) ^ (z))
    P( A, B, C, D,  5,  4, 0xFFFA3942 );
    P( D, A, B, C,  8, 11, 0x8771F681 );
    P( C, D, A, B, 11, 16, 0x6D9D6122 );
    P( B, C, D, A, 14, 23, 0xFDE5380C );
    P( A, B, C, D,  1,  4, 0xA4BEEA44 );
    P( D, A, B, C,  4, 11, 0x4BDECFA9 );
    P( C, D, A, B,  7, 16, 0xF6BB4B60 );
    P( B, C, D, A, 10, 23, 0xBEBFBC70 );
    P( A, B, C, D, 13,  4, 0x289B7EC6 );
    P( D, A, B, C,  0, 11, 0xEAA127FA );
    P( C, D, A, B,  3, 16, 0xD4EF3085 );
    P( B, C, D, A,  6, 23, 0x04881D05 );
    P( A, B, C, D,  9,  4, 0xD9D4D039 );
    P( D, A, B, C, 12, 11, 0xE6DB99E5 );
    P( C, D, A, B, 15, 16, 0x1FA27CF8 );
    P( B, C, D, A,  2, 23, 0xC4AC5665 );
#undef F

#define F(x,y,z) ((y) ^ ((x) | ~(z)))
    P( A, B, C, D,  0,  6, 0xF4292244 );
    P( D, A, B, C,  7, 10, 0x432AFF97 );
    P( C, D, A, B, 14, 15, 0xAB9423A7 );
    P( B, C, D, A,  5, 21, 0xFC93A039 );
    P( A, B, C, D, 12,  6, 0x655B59C3 );
    P( D, A, B, C,  3, 10, 0x8F0CCC92 );
    P( C, D, A, B, 10, 15, 0xFFEFF47D );
    P( B, C, D, A,  1, 21, 0x85845DD1 );
    P( A, B, C, D,  8,  6, 0x6FA87E4F );
    P( D, A, B, C, 15, 10, 0xFE2CE6E0 );
    P( C, D, A, B,  6, 15, 0xA3014314 );
    P( B, C, D, A, 13, 21, 0x4E0811A1 );
    P( A, B, C, D,  4,  6, 0xF7537E82 );
    P( D, A, B, C, 11, 10, 0xBD3AF235 );
    P( C, D, A, B,  2, 15, 0x2AD7D2BB );
    P( B, C, D, A,  9, 21, 0xEB86D391 );
#undef F

    state[0] += A;
    state[1] += B;
    state[2] += C;
    state[3] += D;

    output[0] = state[0];
    output[1] = state[1];
    output[2] = state[2];
    output[3] = state[3];
}

void md5_hash_m1_normal(uint32_t* m, uint32_t output[4], uint32_t s[4]) {
    // init
    uint32_t state[4];
    for (int i = 0; i < 4; i++) {
        state[i] = s[i];
    }

    uint32_t A;
    uint32_t B;
    uint32_t C;
    uint32_t D;

    uint32_t AOld;
    uint32_t BOld;
    uint32_t COld;
    uint32_t DOld;

    uint32_t ANew;
    uint32_t BNew;
    uint32_t CNew;
    uint32_t DNew;


#define S(x,n)                                                          \
            ( ( (x) << (n) ) | ( ( (x) & 0xFFFFFFFF) >> ( 32 - (n) ) ) )

#define P(a,b,c,d,k,s,t)                                                \
            do                                                                  \
            {                                                                   \
                (a) += F((b),(c),(d)) + m[(k)] + (t);                     \
                (a) = S((a),(s)) + (b);                                         \
            } while( 0 )

    A = state[0];
    B = state[1];
    C = state[2];
    D = state[3];

    // 16 kroków, jedno F, jeden cykl
#define F(x,y,z) ((z) ^ ((x) & ((y) ^ (z))))
    AOld = A;
    P( A, B, C, D,  0,  7, 0xD76AA478 );
    ANew = (A & 0x71dff7df) | 0x84200000;
    // after applying changes
    if (A != ANew) { // need to compute m1'
        m[0] = ((((ANew - B)) >> (7)) | ((((ANew - B) ) & 0xFFFFFFFF) << (32 - (7))))
               - AOld - F(B, C, D) - 0xD76AA478;
        A = ANew;
    }

    DOld = D;
    P( D, A, B, C,  1, 12, 0xE8C7B756 );

    DNew = (D & 0x1c06719) | 0x8c000800 | (A & 0x701f10c0);
    if (D != DNew) { // need to compute m1'
        m[1] = ((((DNew - A)) >> (12)) | ((((DNew - A) ) & 0xFFFFFFFF) << (32 - (12))))
               - DOld - F(A, B, C) - 0xE8C7B756;
        D = DNew;
    }

    COld = C;
    P( C, D, A, B,  2, 17, 0x242070DB );

    CNew = (C & 0x1c0e601) | 0xbe1f0966 | (D & 0x00000018);
    if (C != CNew) { // need to compute m1'
        m[2] = ((((CNew - D)) >> (17)) | ((((CNew - D) ) & 0xFFFFFFFF) << (32 - (17))))
               - COld - F(D, A, B) - 0x242070DB;
        C = CNew;
    }

    BOld = B;
    P( B, C, D, A,  3, 22, 0xC1BDCEEE );

    BNew = (B & 0x1c0e000) | 0xba040010 | (C & 0x00000601);
    if (B != BNew) { // need to compute m1'
        m[3] = ((((BNew - C)) >> (22)) | ((((BNew - C) ) & 0xFFFFFFFF) << (32 - (22))))
               - BOld - F(C, D, A) - 0xC1BDCEEE;
        B = BNew;
    }

    AOld = A;
    P( A, B, C, D,  4,  7, 0xF57C0FAF );

    ANew = (A & 0x3c0e000) | 0x482f0e50;
    if (A != ANew) { // need to compute m1'
        m[4] = ((((ANew - B)) >> (7)) | ((((ANew - B) ) & 0xFFFFFFFF) << (32 - (7))))
               - AOld - F(B, C, D) - 0xF57C0FAF;
        A = ANew;
    }

    DOld = D;
    P( D, A, B, C,  5, 12, 0x4787C62A );

    DNew = (D & 0x61cce000) | 0x04220c56;
    if (D != DNew) { // need to compute m1'
        m[5] = ((((DNew - A)) >> (12)) | ((((DNew - A) ) & 0xFFFFFFFF) << (32 - (12))))
               - DOld - F(A, B, C) - 0x4787C62A;
        D = DNew;
    }

    COld = C;
    P( C, D, A, B,  6, 17, 0xA8304613 );

    CNew = (C & 0x604c603e) | 0x96011e01 | (D & 0x01808000);
    if (C != CNew) { // need to compute m1'
        m[6] = ((((CNew - D)) >> (17)) | ((((CNew - D) ) & 0xFFFFFFFF) << (32 - (17))))
               - COld - F(D, A, B) - 0xA8304613;
        C = CNew;
    }

    BOld = B;
    P( B, C, D, A,  7, 22, 0xFD469501 );

    BNew = (B & 0x604c7c3c) | 0x843283c0 | (C & 0x00000002);
    if (B != BNew) { // need to compute m1'
        m[7] = ((((BNew - C)) >> (22)) | ((((BNew - C) ) & 0xFFFFFFFF) << (32 - (22))))
               - BOld - F(C, D, A) - 0xFD469501;
        B = BNew;
    }

    AOld = A;
    P( A, B, C, D,  8,  7, 0x698098D8 );
    ANew = (A & 0x607c6c3c) | 0x9c0101c1 | (B & 0x00001000);
    if (A != ANew) { // need to compute m1'
        m[8] = ((((ANew - B)) >> (7)) | ((((ANew - B) ) & 0xFFFFFFFF) << (32 - (7))))
               - AOld - F(B, C, D) - 0x698098D8;
        A = ANew;
    }

    DOld = D;
    P( D, A, B, C,  9, 12, 0x8B44F7AF );
    DNew = (D &  0x78786c3c) | 0x878383c0;
    if (D != DNew) { // need to compute m1'
        m[9] = ((((DNew - A)) >> (12)) | ((((DNew - A) ) & 0xFFFFFFFF) << (32 - (12))))
               - DOld - F(A, B, C) - 0x8B44F7AF;
        D = DNew;
    }

    COld = C;
    P( C, D, A, B, 10, 17, 0xFFFF5BB1 );

    CNew = (C & 0x7ff00c3c) | 0x800583c3 | (D & 0x86000);
    if (C != CNew) { // need to compute m1'
        m[10] = ((((CNew - D)) >> (17)) | ((((CNew - D) ) & 0xFFFFFFFF) << (32 - (17))))
                - COld - F(D, A, B) - 0xFFFF5BB1;
        C = CNew;
    }

    BOld = B;
    P( B, C, D, A, 11, 22, 0x895CD7BE );

    BNew = (B & 0xf00f7f) | 0x80081080 | (C & 0x7f000000);
    if (B != BNew) { // need to compute m1'
        m[11] = ((((BNew - C)) >> (22)) | ((((BNew - C) ) & 0xFFFFFFFF) << (32 - (22))))
                - BOld - F(C, D, A) - 0x895CD7BE;
        B = BNew;
    }

    AOld = A;
    P( A, B, C, D, 12,  7, 0x6B901122 );

    ANew = (A & 0xf01f77) | 0x3f0fe008;
    if (A != ANew) { // need to compute m1'
        m[12] = ((((ANew - B)) >> (7)) | ((((ANew - B) ) & 0xFFFFFFFF) << (32 - (7))))
                - AOld - F(B, C, D) - 0x6B901122;
        A = ANew;
    }

    DOld = D;
    P( D, A, B, C, 13, 12, 0xFD987193 );

    DNew = (D & 0xf01f77) | 0x400be088;
    if (D != DNew) { // need to compute m1'
        m[13] = ((((DNew - A)) >> (12)) | ((((DNew - A) ) & 0xFFFFFFFF) << (32 - (12))))
                - DOld - F(A, B, C) - 0xFD987193;
        D = DNew;
    }

    COld = C;
    P( C, D, A, B, 14, 17, 0xA679438E );

    CNew = (C & 0xff7ff7) | 0x7d000000;
    if (C != CNew) { // need to compute m1'
        m[14] = ((((CNew - D)) >> (17)) | ((((CNew - D) ) & 0xFFFFFFFF) << (32 - (17))))
                - COld - F(D, A, B) - 0xA679438E;
        C = CNew;
    }

    BOld = B;
    P( B, C, D, A, 15, 22, 0x49B40821 );

    BNew = (B & 0x5fffffff) | 0x20000000;
    if (B != BNew) { // need to compute m1'
        m[15] = ((((BNew - C)) >> (22)) | ((((BNew - C) ) & 0xFFFFFFFF) << (32 - (22))))
                - BOld - F(C, D, A) - 0x49B40821;
        B = BNew;
    }
#undef F

#define F(x,y,z) ((y) ^ ((z) & ((x) ^ (y))))
    P( A, B, C, D,  1,  5, 0xF61E2562 );
    P( D, A, B, C,  6,  9, 0xC040B340 );
    P( C, D, A, B, 11, 14, 0x265E5A51 );
    P( B, C, D, A,  0, 20, 0xE9B6C7AA );
    P( A, B, C, D,  5,  5, 0xD62F105D );
    P( D, A, B, C, 10,  9, 0x02441453 );
    P( C, D, A, B, 15, 14, 0xD8A1E681 );
    P( B, C, D, A,  4, 20, 0xE7D3FBC8 );
    P( A, B, C, D,  9,  5, 0x21E1CDE6 );
    P( D, A, B, C, 14,  9, 0xC33707D6 );
    P( C, D, A, B,  3, 14, 0xF4D50D87 );
    P( B, C, D, A,  8, 20, 0x455A14ED );
    P( A, B, C, D, 13,  5, 0xA9E3E905 );
    P( D, A, B, C,  2,  9, 0xFCEFA3F8 );
    P( C, D, A, B,  7, 14, 0x676F02D9 );
    P( B, C, D, A, 12, 20, 0x8D2A4C8A );

#undef F

#define F(x,y,z) ((x) ^ (y) ^ (z))
    P( A, B, C, D,  5,  4, 0xFFFA3942 );
    P( D, A, B, C,  8, 11, 0x8771F681 );
    P( C, D, A, B, 11, 16, 0x6D9D6122 );
    P( B, C, D, A, 14, 23, 0xFDE5380C );
    P( A, B, C, D,  1,  4, 0xA4BEEA44 );
    P( D, A, B, C,  4, 11, 0x4BDECFA9 );
    P( C, D, A, B,  7, 16, 0xF6BB4B60 );
    P( B, C, D, A, 10, 23, 0xBEBFBC70 );
    P( A, B, C, D, 13,  4, 0x289B7EC6 );
    P( D, A, B, C,  0, 11, 0xEAA127FA );
    P( C, D, A, B,  3, 16, 0xD4EF3085 );
    P( B, C, D, A,  6, 23, 0x04881D05 );
    P( A, B, C, D,  9,  4, 0xD9D4D039 );
    P( D, A, B, C, 12, 11, 0xE6DB99E5 );
    P( C, D, A, B, 15, 16, 0x1FA27CF8 );
    P( B, C, D, A,  2, 23, 0xC4AC5665 );
#undef F

#define F(x,y,z) ((y) ^ ((x) | ~(z)))
    P( A, B, C, D,  0,  6, 0xF4292244 );
    P( D, A, B, C,  7, 10, 0x432AFF97 );
    P( C, D, A, B, 14, 15, 0xAB9423A7 );
    P( B, C, D, A,  5, 21, 0xFC93A039 );
    P( A, B, C, D, 12,  6, 0x655B59C3 );
    P( D, A, B, C,  3, 10, 0x8F0CCC92 );
    P( C, D, A, B, 10, 15, 0xFFEFF47D );
    P( B, C, D, A,  1, 21, 0x85845DD1 );
    P( A, B, C, D,  8,  6, 0x6FA87E4F );
    P( D, A, B, C, 15, 10, 0xFE2CE6E0 );
    P( C, D, A, B,  6, 15, 0xA3014314 );
    P( B, C, D, A, 13, 21, 0x4E0811A1 );
    P( A, B, C, D,  4,  6, 0xF7537E82 );
    P( D, A, B, C, 11, 10, 0xBD3AF235 );
    P( C, D, A, B,  2, 15, 0x2AD7D2BB );
    P( B, C, D, A,  9, 21, 0xEB86D391 );
#undef F

    state[0] += A;
    state[1] += B;
    state[2] += C;
    state[3] += D;

    output[0] = state[0];
    output[1] = state[1];
    output[2] = state[2];
    output[3] = state[3];
}

int main() {
    int size_of_message = 16;
    // N * K + 16 * i
    uint32_t mess1[16];
    uint32_t digest[4];
   // uint32_t* messages = (uint32_t*)malloc(sizeof(uint32_t) * N * K * size_of_message);
   // uint32_t* d_messages;
   // cudaMalloc(&d_messages, sizeof(uint32_t) * N * K * size_of_message);
    int* success = (int*)malloc(sizeof(int) * N * K);
    int* d_success;
    hipMalloc(&d_success, sizeof(int) * N * K);

    clock_t start, end;
    double elapsed;

    uint32_t lastSeed = 0xabcdef;
    for (int i = 0; i < 1024 * 10; i++) {
        printf("Iteration: %d\n", i);
        start = clock();
        // run kernel
        attack<<<N, K>>>(lastSeed, d_success);
       // cudaMemcpy(messages, d_messages, sizeof(uint32_t) * N * K * size_of_message, cudaMemcpyDeviceToHost);
        hipMemcpy(success, d_success, sizeof(int) * N * K, hipMemcpyDeviceToHost);
        // check if collision was found
        for (int j = 0; j < N * K; j++) {
            if (success[j] == 1) { // (j is idx of thread)
                // collision found
                printf("COLLISION FOUND\n");
                // perform experiment but on host with last seed
//                uint32_t s1[] = {0x52589324, 0x3093d7ca, 0x2a06dc54, 0x20c5be06};
//                // m0' hashed (initial state for m1')
//                uint32_t s2[] = {0xd2589324, 0xb293d7ca, 0xac06dc54, 0xa2c5be06};
                uint32_t s1[] = {0xfec19f38, 0xf430a8ea, 0x8ab480e4, 0x58892490};
                uint32_t s2[] = {0x7ec19f38, 0x7630a8ea, 0xcb480e4, 0xda892490};

                uint32_t m1[16];
                uint32_t m1_added[16];
                uint32_t digest1[4];
                uint32_t digest2[4];
                int equal = 0;
                // initialize random generator
                struct ranctx state;
                raninit_normal(&state, lastSeed + j);
                // one thread performs 2^20 experiments
                for (int n = 0; n < 1048576 ; n++) {
                    // get random message m1
                    for (int l = 0; l < 16; l++) {
                        m1[l] = ranval_normal(&state);
                    }

                    // modify and check if this works
                    md5_hash_m1_normal(m1, digest1, s1);

                    // construct m1'
                    for (int l = 0; l < 16; l++) {
                        m1_added[l] = m1[l];
                    }
                    m1_added[4] += 0x80000000;
                    m1_added[11] -= 0x00008000;
                    m1_added[14] += 0x80000000;

                    md5_hash_normal(m1_added, digest2, s2);

                    // check if hashes equal
                    equal = 1;
                    for (int i = 0; i < 4; i++) {
                        if (digest1[i] != digest2[i]) {
                            equal = 0;
                        }
                    }
                    if (equal == 1) { // collision found
                        printf("m1 \n");
                        for (int k = 0; k < 16; k++) {
                            printf("%x ", m1[k]);
                        }
                        printf("\n");
                        printf("m1' \n");
                        for (int k = 0; k < 16; k++) {
                            printf("%x ", m1_added[k]);
                        }
                        printf("\nhash \n");
                        for (int k = 0; k < 4; k++) {
                            printf("%x ", digest1[k]);
                        }
                        printf("\n");
                        return;
                    }
                }
            }

        }
        end = clock();
        elapsed = double(end - start) / CLOCKS_PER_SEC;
        printf("Time: %f, Iteration: %d, Last Seed: %x \n", elapsed, i, lastSeed);
        lastSeed += N * K;
    }

//    cudaFree(d_messages);
    hipFree(d_success);
//    free(messages);
    free(success);
}

/*
 * collision found - iteration 4016, seed  1602a6f + 92*32
 * m1
 * dafde4f7 4bac395a 6bd85fe6 e4685ada 1581f7f8 1d119be8 572d1d2a d1616bd4 9d3f0fb9 cb9bbb79 7d3ffaf6 cf4e1499 c7755cd9 4df3ff84 6d2a53b9 2c72018d
 * m1'
 * dafde4f7 4bac395a 6bd85fe6 e4685ada 9581f7f8 1d119be8 572d1d2a d1616bd4 9d3f0fb9 cb9bbb79 7d3ffaf6 cf4d9499 c7755cd9 4df3ff84 ed2a53b9 2c72018d
 * hash
 * 246da77 60ce90a4 148fc85f fd34275
 */
